#include "hip/hip_runtime.h"
#include "myCuda.cuh"
#include <cstdio>


// define a helper function for checking CUDA errors. See this thread: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace pipcudemo{

	
	// implement the CUDA kernels

	__global__ void add_d(int *a_d, int *b_d, int *c_d){
		if (threadIdx.x == 0){
			*c_d = *a_d + *b_d;
		}
	}

	__global__ void subtract_d(int *a_d, int *b_d, int *c_d){
		if (threadIdx.x == 0){
			*c_d = *a_d - *b_d;
		}
	}


	__global__ void multiply_d(int *a_d, int *b_d, int *c_d){
		if (threadIdx.x == 0){
			*c_d = *a_d * *b_d;
		}
	}

	__global__ void divide_d(int *a_d, int *b_d, int *c_d){
		if (threadIdx.x == 0){
			*c_d = *a_d / *b_d;
		}
	}

	
	// implement the wrappers that copy memory and invoke the kernels
	__host__ int add(int a, int b){
		int *a_d, *b_d, *c_d, result;

		// allocate memory on device
		cudaErrchk(hipMalloc(&a_d, sizeof(int)));
		cudaErrchk(hipMalloc(&b_d, sizeof(int)));
		cudaErrchk(hipMalloc(&c_d, sizeof(int)));

		// copy memory to device
		cudaErrchk(hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice));

		// do the calculation
		add_d<<<1,1>>>(a_d, b_d, c_d);

		// copy result back
		cudaErrchk(hipMemcpy(&result, c_d, sizeof(int), hipMemcpyDeviceToHost));

		return result;
	}

	
	__host__ int subtract(int a, int b){
		int *a_d, *b_d, *c_d, result;

		// allocate memory on device
		cudaErrchk(hipMalloc(&a_d, sizeof(int)));
		cudaErrchk(hipMalloc(&b_d, sizeof(int)));
		cudaErrchk(hipMalloc(&c_d, sizeof(int)));

		// copy memory to device
		cudaErrchk(hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice));

		// do the calculation
		subtract_d<<<1,1>>>(a_d, b_d, c_d);

		// copy result back
		cudaErrchk(hipMemcpy(&result, c_d, sizeof(int), hipMemcpyDeviceToHost));

		return result;
	}

	
	__host__ int multiply(int a, int b){
		int *a_d, *b_d, *c_d, result;

		// allocate memory on device
		cudaErrchk(hipMalloc(&a_d, sizeof(int)));
		cudaErrchk(hipMalloc(&b_d, sizeof(int)));
		cudaErrchk(hipMalloc(&c_d, sizeof(int)));

		// copy memory to device
		cudaErrchk(hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice));

		// do the calculation
		multiply_d<<<1,1>>>(a_d, b_d, c_d);

		// copy result back
		cudaErrchk(hipMemcpy(&result, c_d, sizeof(int), hipMemcpyDeviceToHost));

		return result;
	}

	
	__host__ int divide(int a, int b){
		int *a_d, *b_d, *c_d, result;

		// allocate memory on device
		cudaErrchk(hipMalloc(&a_d, sizeof(int)));
		cudaErrchk(hipMalloc(&b_d, sizeof(int)));
		cudaErrchk(hipMalloc(&c_d, sizeof(int)));

		// copy memory to device
		cudaErrchk(hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice));

		// do the calculation
		divide_d<<<1,1>>>(a_d, b_d, c_d);

		// copy result back
		cudaErrchk(hipMemcpy(&result, c_d, sizeof(int), hipMemcpyDeviceToHost));

		return result;
	}

}