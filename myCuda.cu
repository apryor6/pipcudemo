#include "hip/hip_runtime.h"
#include "myCuda.cuh"
#include <cstdio>

#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace myCuda{

	 
	__global__ void add_d(int *a_d, int *b_d, int *c_d){
		if (threadIdx.x == 0){
			*c_d = *a_d + *b_d;
		}
	}

	
	int add(int a, int b){
		int *a_d, *b_d, *c_d, result;
		cudaErrchk(hipMalloc(&a_d, sizeof(int)));
		cudaErrchk(hipMalloc(&b_d, sizeof(int)));
		cudaErrchk(hipMalloc(&c_d, sizeof(int)));
		cudaErrchk(hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice));
		cudaErrchk(hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice));
		add_d<<<1,1>>>(a_d, b_d, c_d);
		cudaErrchk(hipMemcpy(&result, c_d, sizeof(int), hipMemcpyDeviceToHost));

		return result;
	}

	
	T subtract(T a, T b){
	}

	
	T multiply(T a, T b){
	}

	
	T divide(T a, T b){
	}


}